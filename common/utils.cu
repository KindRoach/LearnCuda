#include "utils.cuh"
#include <iostream>
#include <chrono>

void TIME(std::function<void(void)> func) {
    using std::chrono::high_resolution_clock;
    using std::chrono::duration_cast;
    using std::chrono::duration;
    using std::chrono::milliseconds;

    auto t1 = high_resolution_clock::now();
    func();
    auto t2 = high_resolution_clock::now();

    duration<double, std::milli> ms_double = t2 - t1;
    std::cout << ms_double.count() << "ms" << std::endl;
}