//
// Created by kindr on 2021/5/3.
//

#include "arrayHelper.cuh"
#include "vectorHelper.cuh"
#include "utils.cuh"

void addArrayOnCPU(const float *A, const float *B, float *C, const size_t N) {
    for (size_t i = 0; i < N; ++i) {
        C[i] = A[i] + B[i];
    }
}

bool isFloatArraySame(const float *A, const float *B, const size_t N, float error) {
    for (int i = 0; i < N; ++i) {
        if (!isFloatSame(A[i], B[i], error)) {
            return false;
        }
    }
    return true;
}

void randomInitArray(float *vec, const size_t N) {
    const int threadNum = 1024;

    hiprandState *d_random;
    hipMalloc(&d_random, threadNum * sizeof(hiprandState));
    init_random<<<1, threadNum>>>(d_random);
    CHECK(hipGetLastError())

    float *d_vec;
    size_t nBytes = N * sizeof(float);
    hipMalloc(&d_vec, nBytes);
    CHECK(hipGetLastError())

    gpu_random<<<1, threadNum>>>(d_random, d_vec, N);
    hipDeviceSynchronize();
    CHECK(hipGetLastError())

    hipMemcpy(vec, d_vec, nBytes, hipMemcpyDeviceToHost);
    hipFree(d_vec);
    hipFree(d_random);
    CHECK(hipGetLastError())
}
