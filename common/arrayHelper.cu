//
// Created by kindr on 2021/5/3.
//

#include "arrayHelper.cuh"
#include "vectorHelper.cuh"
#include "utils.cuh"

bool isFloatArraySame(const float *A, const float *&B, const size_t N, float error) {
    for (int i = 0; i < N; ++i) {
        if (!isFloatSame(A[i], B[i], error)) {
            return false;
        }
    }
    return true;
}

void randomInitArray(float *vec, const size_t N) {
    const int threadNum = 1024;

    hiprandState *d_random;
    hipMalloc(&d_random, threadNum * sizeof(hiprandState));
    init_random<<<1, threadNum>>>(d_random);
    CHECK(hipGetLastError())

    float *d_vec;
    size_t nBytes = N * sizeof(float);
    hipMalloc(&d_vec, nBytes);
    CHECK(hipGetLastError())

    gpu_random<<<1, threadNum>>>(d_random, d_vec, N);
    hipDeviceSynchronize();
    CHECK(hipGetLastError())

    hipMemcpy(vec, d_vec, nBytes, hipMemcpyDeviceToHost);
    hipFree(d_vec);
    hipFree(d_random);
    CHECK(hipGetLastError())
}
