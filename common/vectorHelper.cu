#include "vectorHelper.cuh"
#include "utils.cuh"
#include "randomHelper.cuh"

#include <vector>

void addVectorOnCPU(const std::vector<float> &A, const std::vector<float> &B, std::vector<float> &C) {
    for (size_t i = 0; i < C.size(); ++i) {
        C[i] = A[i] + B[i];
    }
}

bool isFloatSame(float a, float b, float error) {
    return a - b < error && b - a < error;
}

bool isFloatVectorSame(const std::vector<float> &A, const std::vector<float> &B, float error) {
    for (int i = 0; i < A.size(); ++i) {
        if (!isFloatSame(A[i], B[i], error)) {
            return false;
        }
    }
    return true;
}

void randomInitVector(std::vector<float> &vec) {
    const int threadNum = 1024;

    hiprandState *d_random;
    hipMalloc(&d_random, threadNum * sizeof(hiprandState));
    init_random<<<1, threadNum>>>(d_random);
    CHECK(hipGetLastError())

    float *d_vec;
    size_t nBytes = vec.size() * sizeof(float);
    hipMalloc(&d_vec, nBytes);
    CHECK(hipGetLastError())

    gpu_random<<<1, threadNum>>>(d_random, d_vec, vec.size());
    hipDeviceSynchronize();
    CHECK(hipGetLastError())

    hipMemcpy(vec.data(), d_vec, nBytes, hipMemcpyDeviceToHost);
    hipFree(d_vec);
    hipFree(d_random);
    CHECK(hipGetLastError())
}
