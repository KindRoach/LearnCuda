#include "hip/hip_runtime.h"
#include "vectorHelper.cuh"
#include "utils.cuh"

#include <vector>
#include <hiprand/hiprand_kernel.h>

bool isFloatSame(float a, float b, float error) {
    return a - b < error && b - a < error;
}

bool isFloatVectorSame(const std::vector<float> &A, const std::vector<float> &B, float error) {
    for (int i = 0; i < A.size(); ++i) {
        if (!isFloatSame(A[i], B[i], error)) {
            return false;
        }
    }
    return true;
}

__global__ void init_random(hiprandState *state) {
    auto idx = threadIdx.x;
    hiprand_init(idx, idx, 0, &state[idx]);
}

__global__ void gpu_random(hiprandState *states, float *d_vec, size_t N) {
    for (size_t i = 0; i < N; i += blockDim.x) {
        size_t idx = i + threadIdx.x;
        if (idx < N) {
            float uniform = hiprand_uniform(&states[threadIdx.x]);
            d_vec[idx] = (uniform - 0.5f) * 2;
        }
    }
}

__global__ void gpu_random(hiprandState *states, int *d_vec, const size_t N) {
    for (size_t i = 0; i < N; i += blockDim.x) {
        size_t idx = i + threadIdx.x;
        if (idx < N) {
            float uniform = hiprand_uniform(&states[threadIdx.x]);
            d_vec[idx] = static_cast<int>((uniform - 0.5) * 1e4);
        }
    }
}

void randomInitVector(std::vector<float> &vec) {
    const int threadNum = 1024;

    hiprandState *d_random;
    hipMalloc(&d_random, threadNum * sizeof(hiprandState));
    init_random<<<1, threadNum>>>(d_random);
    CHECK(hipGetLastError())

    float *d_vec;
    size_t nBytes = vec.size() * sizeof(float);
    hipMalloc(&d_vec, nBytes);
    CHECK(hipGetLastError())

    gpu_random<<<1, threadNum>>>(d_random, d_vec, vec.size());
    hipDeviceSynchronize();
    CHECK(hipGetLastError())

    hipMemcpy(vec.data(), d_vec, nBytes, hipMemcpyDeviceToHost);
    hipFree(d_vec);
    hipFree(d_random);
    CHECK(hipGetLastError())
}
