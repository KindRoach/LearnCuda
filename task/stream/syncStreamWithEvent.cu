#include "hip/hip_runtime.h"
//
// Created by kindr on 2021/5/10.
//

#include "syncStreamWithEvent.cuh"
#include "../../common/utils.cuh"

const int N = 1 << 25;

__global__
void kernel_func() {
    double sum = 0;
    for (int i = 0; i < N; i++) sum += tan(0.1) * tan(0.1);
    printf("sum=%g\n", sum);
}

void syncStreamWithEvent() {
    hipStream_t s1, s2;
    hipStreamCreate(&s1);
    hipStreamCreate(&s2);

    hipEvent_t e;
    hipEventCreate(&e);

    kernel_func<<<1, 1, 0, s1>>>();
    kernel_func<<<1, 1, 0, s1>>>();
    kernel_func<<<1, 1, 0, s2>>>();

    hipEventRecord(e, s1);
    hipStreamWaitEvent(s2, e);

    kernel_func<<<1, 1, 0, s2>>>();

    hipDeviceSynchronize();
    CHECK(hipGetLastError());

    hipEventDestroy(e);
    hipStreamDestroy(s1);
    hipStreamDestroy(s2);
}
