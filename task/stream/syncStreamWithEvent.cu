//
// Created by kindr on 2021/5/10.
//

#include "syncStreamWithEvent.cuh"
#include "../../common/utils.cuh"
#include "multiKernelConcurrent.cuh"

const int N = 1 << 25;

void syncStreamWithEvent() {
    hipStream_t s1, s2;
    hipStreamCreate(&s1);
    hipStreamCreate(&s2);

    hipEvent_t e;
    hipEventCreate(&e);

    math_kernel1<<<1, 1, 0, s1>>>(N);
    math_kernel1<<<1, 1, 0, s1>>>(N);
    math_kernel1<<<1, 1, 0, s2>>>(N);

    hipEventRecord(e, s1);
    hipStreamWaitEvent(s2, e);

    math_kernel1<<<1, 1, 0, s2>>>(N);

    hipDeviceSynchronize();
    CHECK(hipGetLastError());

    hipEventDestroy(e);
    hipStreamDestroy(s1);
    hipStreamDestroy(s2);
}
