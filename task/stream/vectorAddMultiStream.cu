#include "hip/hip_runtime.h"
//
// Created by kindr on 2021/5/10.
//

#include <iostream>
#include "vectorAddMultiStream.cuh"
#include "../../common/arrayHelper.cuh"
#include "../../common/utils.cuh"

const int times = 300;

__global__
void addVectorOnGPU_1M(const float *A, const float *B, float *C, size_t N) {
    unsigned idx = blockDim.x * blockIdx.x + threadIdx.x;
    for (int i = 0; i < times; ++i) if (idx < N) C[idx] = A[idx] + B[idx];
}


void vectorAddMultiStream(size_t nElement, size_t nThread) {
    size_t nBytes = nElement * sizeof(float);

    float *A, *B, *cpuResult, *gpuResult;
    hipHostAlloc(&A, nBytes, hipHostMallocDefault);
    hipHostAlloc(&B, nBytes, hipHostMallocDefault);
    hipHostAlloc(&cpuResult, nBytes, hipHostMallocDefault);
    hipHostAlloc(&gpuResult, nBytes, hipHostMallocDefault);


    randomInitArray(A, nElement);
    randomInitArray(B, nElement);

    addArrayOnCPU(A, B, cpuResult, nElement);

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, nBytes);
    hipMalloc(&d_B, nBytes);
    hipMalloc(&d_C, nBytes);
    CHECK(hipGetLastError())


    const int nStream = 4;
    size_t nStreamBytes = nStream * sizeof(hipStream_t);
    auto *stream = static_cast<hipStream_t *>(malloc(nStreamBytes));
    for (int i = 0; i < nStream; i++) {
        hipStreamCreate(&stream[i]);
    }
    CHECK(hipGetLastError());

    size_t nElementPerStream = nElement / nStream;
    size_t nBytesPerStream = nElementPerStream * sizeof(float);
    size_t nBlockPerStream = (nElementPerStream + nThread - 1) / nThread;
    for (int i = 0; i < nStream; i++) {
        size_t offset = i * nElementPerStream;
        hipMemcpyAsync(&d_A[offset], &A[offset], nBytesPerStream, hipMemcpyHostToDevice, stream[i]);
        hipMemcpyAsync(&d_B[offset], &B[offset], nBytesPerStream, hipMemcpyHostToDevice, stream[i]);
        addVectorOnGPU_1M<<< nBlockPerStream, nThread, 0, stream[i] >>>(
                &d_A[offset], &d_B[offset], &d_C[offset], nElementPerStream);
        hipMemcpyAsync(&gpuResult[offset], &d_C[offset], nBytesPerStream, hipMemcpyDeviceToHost, stream[i]);
    }

    hipDeviceSynchronize();
    CHECK(hipGetLastError());

    for (int i = 0; i < nStream; i++) {
        hipStreamDestroy(stream[i]);
    }

    std::cout << std::boolalpha << "Is same?: "
              << isFloatArraySame(cpuResult, gpuResult, nElement, 1e-8)
              << std::endl;

    hipHostFree(A);
    hipHostFree(B);
    hipHostFree(cpuResult);
    hipHostFree(gpuResult);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
