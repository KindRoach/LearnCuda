//
// Created by kindr on 2021/5/8.
//

#include "multiKernelConcurrent.cuh"
#include "../../common/utils.cuh"
#include "../memory/zeroCopyMemory.cuh"
#include <cstdio>

void multiKernelConcurrent(size_t nElement, size_t nThread) {
    float *vec, *d_vec;
    size_t nBytes = nElement * sizeof(float);
    vec = static_cast<float *>(malloc(nBytes));
    hipMalloc(&d_vec, nBytes);
    hipMemset(d_vec, 0, nBytes);
    CHECK(hipGetLastError());

    int n_stream = 16;
    const size_t &nStreamBytes = n_stream * sizeof(hipStream_t);
    auto *stream = static_cast<hipStream_t *>(malloc(nStreamBytes));
    for (int i = 0; i < n_stream; i++) {
        hipStreamCreate(&stream[i]);
    }
    CHECK(hipGetLastError());

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, nullptr);
    CHECK(hipGetLastError());

    size_t nBlock = (nElement + nThread - 1) / nThread;
//    for (int i = 0; i < n_stream; i++) {
//        addOne<<<nBlock, nThread, 0, stream[i]>>>(vec, nElement);
//        addOne<<<nBlock, nThread, 0, stream[i]>>>(vec, nElement);
//        addOne<<<nBlock, nThread, 0, stream[i]>>>(vec, nElement);
//        addOne<<<nBlock, nThread, 0, stream[i]>>>(vec, nElement);
//    }

    for (int i = 0; i < n_stream; i++) {
        addOne<<<nBlock, nThread>>>(d_vec, nElement);
        addOne<<<nBlock, nThread>>>(d_vec, nElement);
        addOne<<<nBlock, nThread>>>(d_vec, nElement);
        addOne<<<nBlock, nThread>>>(d_vec, nElement);
    }
    CHECK(hipGetLastError());


    hipEventRecord(stop, nullptr);
    hipEventSynchronize(stop);
    CHECK(hipGetLastError());

    float elapsed_time;
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf("elapsed time:%f ms\n", elapsed_time);

    for (int i = 0; i < n_stream; i++) {
        hipStreamDestroy(stream[i]);
    }


    hipMemcpy(vec, d_vec, nBytes, hipMemcpyDeviceToHost);
    bool isSame = true;
    for (size_t i = 0; i < nElement; ++i) {
        if (vec[i] != 4.f * n_stream) {
            isSame = false;
        }
    }

    printf("isSame?: %s", isSame ? "true" : "false");

    hipHostFree(vec);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    free(stream);
}
