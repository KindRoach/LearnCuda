//
// Created by kindr on 2021/5/12.
//

#include "graphConcurrent.cuh"
#include "multiKernelConcurrent.cuh"

const int N = 1 << 25;

void graphConcurrent() {
    hipStream_t s1, s2;
    hipStreamCreate(&s1);
    hipStreamCreate(&s2);

    // 开始捕获流操作
    hipStreamBeginCapture(s1, hipStreamCaptureModeGlobal);

    math_kernel1<<<1, 1, 0, s1>>>(N);

    hipEvent_t e1, e2;
    hipEventCreate(&e1);
    hipEventCreate(&e2);

    hipEventRecord(e1, s1);
    hipStreamWaitEvent(s2, e1);

    math_kernel2<<<1, 1, 0, s1>>>(N);
    math_kernel2<<<1, 1, 0, s2>>>(N);

    hipEventRecord(e2, s2);
    hipStreamWaitEvent(s1, e2);

    math_kernel1<<<1, 1, 0, s1>>>(N);

    // 捕获结束
    hipGraph_t graph;
    hipStreamEndCapture(s1, &graph);

    hipGraphExec_t graphExec;
    hipGraphInstantiate(&graphExec, graph, nullptr, nullptr, 0);
    for (int i = 0; i < 2; i++) {
        hipGraphLaunch(graphExec, nullptr);
    }

    hipDeviceSynchronize();
}
