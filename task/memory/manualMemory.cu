//
// Created by kindr on 2021/4/28.
//

#include "manualMemory.cuh"
#include "../../common/utils.cuh"
#include "zeroCopyMemory.cuh"

#include <cstdio>

void manualMemory(size_t nElement, size_t nThread) {

    size_t nBytes = nElement * sizeof(float);
    auto *vec = (float *) malloc(nBytes);
    memset(vec, 0, nBytes);

    float *d_vec;
    hipMalloc(&d_vec, nBytes);
    CHECK(hipGetLastError());

    hipMemcpy(d_vec, vec, nBytes, hipMemcpyHostToDevice);
    CHECK(hipGetLastError());

    size_t nBlock = (nElement + nThread - 1) / nThread;
    addOne<<<nBlock, nThread>>>(d_vec, nElement);
    hipDeviceSynchronize();
    CHECK(hipGetLastError());

    hipMemcpy(vec, d_vec, nBytes, hipMemcpyDeviceToHost);
    CHECK(hipGetLastError());

    bool isSame = true;
    for (size_t i = 0; i < nElement; ++i) {
        if (vec[i] != 1.f) {
            isSame = false;
        }
    }

    printf("isSame?: %s", isSame ? "true" : "false");

    hipFree(d_vec);
    free(vec);
}
