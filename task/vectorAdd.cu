#include "hip/hip_runtime.h"
#include "vectorAdd.cuh"
#include "../common/utils.cuh"
#include "../common/vectorHelper.cuh"

#include <iostream>

void addVectorOnCPU(const std::vector<float> &A, const std::vector<float> &B, std::vector<float> &C) {
    for (size_t i = 0; i < C.size(); ++i) {
        C[i] = A[i] + B[i];
    }
}

__global__
void addVectorOnGPU(const float *A, const float *B, float *C, size_t N) {
    unsigned idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < N) C[idx] = A[idx] + B[idx];
}

void performVectorAdd(size_t nElement, size_t nThread) {
    auto A = std::vector<float>(nElement);
    auto B = std::vector<float>(nElement);
    auto cpuResult = std::vector<float>(nElement);
    auto gpuResult = std::vector<float>(nElement);

    randomInitVector(A);
    randomInitVector(B);

    std::cout << "addVectorOnCPU:";
    TIME([&]() {
        addVectorOnCPU(A, B, cpuResult);
    });

    size_t nBytes = nElement * sizeof(float);
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, nBytes);
    hipMalloc(&d_B, nBytes);
    hipMalloc(&d_C, nBytes);
    CHECK(hipGetLastError())

    std::cout << "addVectorOnGPU:";
    TIME([&]() {
        hipMemcpy(d_A, A.data(), nBytes, hipMemcpyHostToDevice);
        hipMemcpy(d_B, B.data(), nBytes, hipMemcpyHostToDevice);
        CHECK(hipGetLastError())

        size_t nBlock = (nElement + nThread - 1) / nThread;
        addVectorOnGPU<<<nBlock, nThread>>>(d_A, d_B, d_C, nElement);
        hipDeviceSynchronize();
        CHECK(hipGetLastError())

        hipMemcpy(gpuResult.data(), d_C, nBytes, hipMemcpyDeviceToHost);
        CHECK(hipGetLastError())
    });

    std::cout << std::boolalpha << "Is same?: "
              << isFloatVectorSame(cpuResult, gpuResult)
              << std::endl;

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
