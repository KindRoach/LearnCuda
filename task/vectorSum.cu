#include "hip/hip_runtime.h"
#include <numeric>
#include <iostream>
#include "../common/utils.cuh"
#include "../common/vectorHelper.cuh"

__global__ void sumVectorOnGPU_Ver1(float *vec) {
    unsigned offset = blockDim.x * blockIdx.x;
    for (unsigned i = 1; i < blockDim.x; i <<= 1) {
        if (threadIdx.x % (i << 1) == 0) {
            unsigned idx = offset + threadIdx.x;
            vec[idx] = vec[idx] + vec[idx + i];
        }
        __syncthreads();
    }
}

__global__ void sumVectorOnGPU_Ver2(float *vec) {
    unsigned offset = blockDim.x * blockIdx.x;
    for (unsigned i = 1; i < blockDim.x; i <<= 1) {
        if (threadIdx.x < blockDim.x / (i << 1)) {
            unsigned idx = offset + threadIdx.x * (i << 1);
            vec[idx] = vec[idx] + vec[idx + i];
        }
        __syncthreads();
    }
}

__global__ void sumVectorOnGPU_Ver3(float *vec) {
    unsigned offset = blockDim.x * blockIdx.x;
    for (unsigned i = blockDim.x >> 1; i > 0; i >>= 1) {
        if (threadIdx.x < i) {
            unsigned idx = offset + threadIdx.x;
            vec[idx] = vec[idx] + vec[idx + i];
        }
        __syncthreads();
    }
}

void performVectorSum(size_t nElement, size_t nThread) {
    auto vec = std::vector<float>(nElement);
    randomInitVector(vec);

    float cpuResult = 0;
    std::cout << "sumVectorOnCPU:";
    TIME([&]() {
        cpuResult = std::accumulate(vec.begin(), vec.end(), 0.f);
    });

    // padding 0
    size_t nBlock = (nElement + nThread - 1) / nThread;
    nElement = nBlock * nThread;
    auto gpuOutput = std::vector<float>(nElement);
    for (size_t i = vec.size(); i < nElement; ++i) {
        vec.emplace_back(0.f);
    }

    size_t nBytes = nElement * sizeof(float);

    float *d_vec;
    hipMalloc(&d_vec, nBytes);
    CHECK(hipGetLastError())

    float gpuResult_ver1 = 0;
    std::cout << "sumVectorOnGPU_Ver1:";
    TIME([&]() {
        hipMemcpy(d_vec, vec.data(), nBytes, hipMemcpyHostToDevice);
        sumVectorOnGPU_Ver1<<<nBlock, nThread>>>(d_vec);
        hipDeviceSynchronize();
        CHECK(hipGetLastError())

        hipMemcpy(gpuOutput.data(), d_vec, nBytes, hipMemcpyDeviceToHost);
        CHECK(hipGetLastError())

        gpuResult_ver1 = 0;
        for (size_t i = 0; i < nElement; i += nThread) {
            gpuResult_ver1 += gpuOutput[i];
        }
    });

    float gpuResult_ver2 = 0;
    std::cout << "sumVectorOnGPU_Ver2:";
    TIME([&]() {
        hipMemcpy(d_vec, vec.data(), nBytes, hipMemcpyHostToDevice);
        sumVectorOnGPU_Ver2<<<nBlock, nThread>>>(d_vec);
        hipDeviceSynchronize();
        CHECK(hipGetLastError())

        hipMemcpy(gpuOutput.data(), d_vec, nBytes, hipMemcpyDeviceToHost);
        CHECK(hipGetLastError())

        gpuResult_ver2 = 0;
        for (size_t i = 0; i < nElement; i += nThread) {
            gpuResult_ver2 += gpuOutput[i];
        }

    });

    float gpuResult_ver3 = 0;
    std::cout << "sumVectorOnGPU_Ver2:";
    TIME([&]() {
        hipMemcpy(d_vec, vec.data(), nBytes, hipMemcpyHostToDevice);
        sumVectorOnGPU_Ver3<<<nBlock, nThread>>>(d_vec);
        hipDeviceSynchronize();
        CHECK(hipGetLastError())

        hipMemcpy(gpuOutput.data(), d_vec, nBytes, hipMemcpyDeviceToHost);
        CHECK(hipGetLastError())

        gpuResult_ver3 = 0;
        for (size_t i = 0; i < nElement; i += nThread) {
            gpuResult_ver3 += gpuOutput[i];
        }

    });

    printf("cpuResult: %f\ngpuResult_ver1: %f\ngpuResult_ver2: %f\ngpuResult_ver3: %f\n",
           cpuResult, gpuResult_ver1, gpuResult_ver2, gpuResult_ver3);

    hipFree(d_vec);
}


