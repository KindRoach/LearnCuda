#include "hip/hip_runtime.h"
#include <numeric>
#include <iostream>
#include "../common/utils.cuh"
#include "../common/vectorHelper.cuh"

__global__ void sumVectorOnGPU(int *vec) {
    unsigned offset = blockDim.x * blockIdx.x;
    for (unsigned i = 1; i < blockDim.x; i <<= 1) {
        if (threadIdx.x % (i << 1) == 0) {
            unsigned idx = offset + threadIdx.x;
            vec[idx] = vec[idx] + vec[idx + i];
        }
        __syncthreads();
    }
}

void performVectorSum(size_t nElement, size_t nThread) {
    auto vec = std::vector<int>(nElement);
    randomInitVector(vec);

    // padding 0
    size_t nBlock = (nElement + nThread - 1) / nThread;
    nElement = nBlock * nThread;
    for (size_t i = vec.size(); i < nElement; ++i) {
        vec.emplace_back(0.f);
    }

    int cpuResult = 0;
    std::cout << "sumVectorOnCPU:";
    TIME([&]() {
        cpuResult = std::accumulate(vec.begin(), vec.end(), 0);
    });

    size_t nBytes = nElement * sizeof(int);

    int *d_vec;
    hipMalloc(&d_vec, nBytes);
    CHECK(hipGetLastError())

    int gpuResult = 0;
    auto gpuOutput = std::vector<int>(nElement);

    std::cout << "sumVectorOnGPU:";
    TIME([&]() {
        hipMemcpy(d_vec, vec.data(), nBytes, hipMemcpyHostToDevice);
        sumVectorOnGPU<<<nBlock, nThread>>>(d_vec);
        hipDeviceSynchronize();
        CHECK(hipGetLastError())

        hipMemcpy(gpuOutput.data(), d_vec, nBytes, hipMemcpyDeviceToHost);
        CHECK(hipGetLastError())

        for (size_t i = 0; i < nElement; i += nThread) {
            gpuResult += gpuOutput[i];
        }
    });

    printf("Is same?: %s (%d==%d)\n",
           cpuResult == gpuResult ? "true" : "false",
           cpuResult, gpuResult);

    hipFree(d_vec);
}


