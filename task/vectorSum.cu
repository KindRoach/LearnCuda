#include "hip/hip_runtime.h"
#include <numeric>
#include <iostream>
#include "../common/utils.cuh"
#include "../common/vectorHelper.cuh"

__global__ void sumVectorOnGPU_Ver1(float *vec, float *res) {
    float *vec_i = vec + blockDim.x * 2 * blockIdx.x;
    for (unsigned i = 1; i <= blockDim.x; i *= 2) {
        if (threadIdx.x % i == 0) {
            unsigned idx = threadIdx.x * 2;
            vec_i[idx] = vec_i[idx] + vec_i[idx + i];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) res[blockIdx.x] = vec_i[0];
}

__global__ void sumVectorOnGPU_Ver2(float *vec, float *res) {
    float *vec_i = vec + blockDim.x * 2 * blockIdx.x;
    for (unsigned i = 1; i <= blockDim.x; i *= 2) {
        if (threadIdx.x < blockDim.x / i) {
            unsigned idx = threadIdx.x * (i << 1);
            vec_i[idx] = vec_i[idx] + vec_i[idx + i];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) res[blockIdx.x] = vec_i[0];
}

__global__ void sumVectorOnGPU_Ver3(float *vec, float *res) {
    float *vec_i = vec + blockDim.x * 2 * blockIdx.x;
    for (unsigned i = blockDim.x; i > 0; i /= 2) {
        if (threadIdx.x < i) {
            unsigned idx = threadIdx.x;
            vec_i[idx] = vec_i[idx] + vec_i[idx + i];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) res[blockIdx.x] = vec_i[0];
}


void performVectorSum(size_t nElement, size_t nThread) {
    auto vec = std::vector<float>(nElement);
    randomInitVector(vec);

    float cpuResult = 0;
    std::cout << "sumVectorOnCPU:";
    TIME([&]() {
        cpuResult = std::accumulate(vec.begin(), vec.end(), 0.f);
        printf(" %f --- ", cpuResult);
    });


    // padding 0 for GPU
    size_t nBlock = (nElement + nThread * 2 - 1) / (nThread * 2);
    nElement = nBlock * nThread * 2;
    for (size_t i = vec.size(); i < nElement; ++i) {
        vec.emplace_back(0.f);
    }

    float *d_vec, *d_result;
    hipMalloc(&d_vec, nElement * sizeof(float));
    hipMalloc(&d_result, nBlock * sizeof(float));
    CHECK(hipGetLastError())

    auto ver1_func = [&]() { sumVectorOnGPU_Ver1<<<nBlock, nThread>>>(d_vec, d_result); };
    auto ver2_func = [&]() { sumVectorOnGPU_Ver2<<<nBlock, nThread>>>(d_vec, d_result); };
    auto ver3_func = [&]() { sumVectorOnGPU_Ver3<<<nBlock, nThread>>>(d_vec, d_result); };
    auto gpuOutput = std::vector<float>(nBlock);

    auto sumVector = [&](auto kernel) {
        TIME([&]() {
            printf("sumVectorOnGPU:");

            hipMemcpy(d_vec, vec.data(), nElement * sizeof(float), hipMemcpyHostToDevice);
            kernel();
            hipDeviceSynchronize();
            CHECK(hipGetLastError())

            hipMemcpy(gpuOutput.data(), d_result, nBlock * sizeof(float), hipMemcpyDeviceToHost);
            CHECK(hipGetLastError())

            float gpuResult = 0;
            for (size_t i = 0; i < nBlock; ++i) {
                gpuResult += gpuOutput[i];
            }

            printf(" %f --- ", gpuResult);
        });
    };

    sumVector(ver1_func);
    sumVector(ver2_func);
    sumVector(ver3_func);

    hipFree(d_vec);
    hipFree(d_result);
}


